#include "hip/hip_runtime.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "utils.hpp"
#include "update.hpp"
#include "memoryManager.hpp"
#include "cuStinger.hpp"

using namespace std;
#include <stdio.h>
#include <string.h>


// void initializeCuStinger(cuStingerConfig);

// __global__ void devInitVertexData(cuStinger* custing,uint8_t* temp)
__global__ void devInitVertexData(cuStinger::cusVertexData *dVD,vertexId_t nv,uint8_t* temp)
{
	if(threadIdx.x!=0 || blockIdx.x!=0)
		DEV_CUSTINGER_ERROR("Number of threads and thread blocks for initializing vertex should always be one");
	// cuStinger::cusVertexData *dVD = custing->dVD;

	dVD->mem = temp;
	int32_t pos=0;

	dVD->adj 		= (cuStinger::cusEdgeData**)(dVD->getMem() + pos); 	pos+=sizeof(cuStinger::cusEdgeData*)*nv;
	dVD->edMem 		= (uint8_t**)(dVD->getMem() + pos); 				pos+=sizeof(uint8_t*)*nv;
	dVD->used 		= (length_t*)(dVD->getMem() + pos); 				pos+=sizeof(length_t)*nv;
	dVD->max        = (length_t*)(dVD->getMem() + pos); 				pos+=sizeof(length_t)*nv;
	dVD->vw         = (vweight_t*)(dVD->getMem() + pos); 				pos+=sizeof(vweight_t)*nv;
	dVD->vt         = (vtype_t*)(dVD->getMem() + pos); 					pos+=sizeof(vtype_t)*nv;
}

void cuStinger::initVertexDataPointers(cuStinger::cusVertexData *dVD, uint8_t* temp){
	// devInitVertexData<<<1,1>>>(	d_cuStinger,temp);
	devInitVertexData<<<1,1>>>(	dVD,nv,temp);
}

__global__ void devInitEdgeData(cuStinger* custing, int verticesPerThreadBlock){
	vertexId_t v_init=blockIdx.x*verticesPerThreadBlock+threadIdx.x;
	length_t nv = custing->getMaxNV();
	for (vertexId_t v_hat=0; v_hat<verticesPerThreadBlock; v_hat+=blockDim.x){
		vertexId_t v=v_init+v_hat;
		if(v>=nv)
			break;

		//epv = edge per vertex
		length_t epv = custing->dVD->getMax()[v];

		int32_t pos=0;
		cuStinger::cusEdgeData *dED = custing->dVD->adj[v];

		dED->mem = custing->dVD->edMem[v];
		dED->dst = (vertexId_t*)(dED->getMem() + pos); 	pos+=sizeof(vertexId_t)*epv;
		dED->ew  = (eweight_t*)(dED->getMem() + pos); 	pos+=sizeof(eweight_t)*epv;
		dED->et  = (etype_t*)(dED->getMem() + pos); 	pos+=sizeof(etype_t)*epv;
		dED->t1  = (timestamp_t*)(dED->getMem() + pos); pos+=sizeof(timestamp_t)*epv;
		dED->t2  = (timestamp_t*)(dED->getMem() + pos); pos+=sizeof(timestamp_t)*epv;
	}
}

void cuStinger::initEdgeDataPointers(){
	dim3 numBlocks(1, 1);
	int32_t threads=64;
	dim3 threadsPerBlock(threads, 1);

	numBlocks.x = ceil((float)nv/(float)threads);
	if (numBlocks.x>16000){
		numBlocks.x=16000;
	}	

	int32_t verticesPerThreadBlock = threads;

	if(numBlocks.x>1)
		 verticesPerThreadBlock = ceil(float(nv)/float(numBlocks.x-1));		

	devInitEdgeData<<<numBlocks,threadsPerBlock>>>(	d_cuStinger,verticesPerThreadBlock);
}

__global__ void devMakeGPUStinger(vertexId_t* d_off, length_t* d_adj,
	int verticesPerThreadBlock,cuStinger* custing){
	length_t* d_utilized = custing->dVD->getUsed();
	length_t* d_max = custing->dVD->getMax();


	int32_t v_init=blockIdx.x*verticesPerThreadBlock;
	for (int v_hat=0; v_hat<verticesPerThreadBlock; v_hat++){
		int32_t v=v_init+v_hat;

		if(v>=custing->nv)
			break;
		cuStinger::cusEdgeData* adjv = custing->dVD->adj[v];
		for(int32_t e=threadIdx.x; e<d_utilized[v]; e+=blockDim.x){
				adjv->dst[e]=d_adj[d_off[v]+e];
		}
		for(int32_t e=threadIdx.x + d_utilized[v]; e < d_max[v]; e+=blockDim.x){
			adjv->dst[e]=DELETION_MARKER;
		}
	}
}

void cuStinger::internalCSRTocuStinger(length_t* h_off, vertexId_t* h_adj, length_t ne){
	length_t* d_off = (length_t*)allocDeviceArray(nv+1,sizeof(length_t));
	vertexId_t* d_adj = (vertexId_t*)allocDeviceArray(ne,sizeof(vertexId_t));
	copyArrayHostToDevice(h_off,d_off,nv+1,sizeof(length_t));
	copyArrayHostToDevice(h_adj,d_adj,ne,sizeof(vertexId_t));

	dim3 numBlocks(1, 1);
	int32_t threads=64;
	dim3 threadsPerBlock(threads, 1);

	numBlocks.x = ceil((float)nv/(float)threads);
	if (numBlocks.x>16000){
		numBlocks.x=16000;
	}	

	int32_t verticesPerThreadBlock;
	if(numBlocks.x==1)
		verticesPerThreadBlock=nv;
	else
		verticesPerThreadBlock= ceil(float(nv)/float(numBlocks.x-1));

	devMakeGPUStinger<<<numBlocks,threadsPerBlock>>>(d_off,d_adj,verticesPerThreadBlock, d_cuStinger);

	freeDeviceArray(d_adj);	
	freeDeviceArray(d_off);
}



#define SUM_BLOCK_SIZE 512
__global__ void devSumArray(length_t * input, length_t * output, length_t len) {
    __shared__ length_t partialSum[2 * SUM_BLOCK_SIZE];
    //Load a segment of the input vector into shared memory
    length_t tid = threadIdx.x, start = 2 * blockIdx.x * SUM_BLOCK_SIZE;
    if (start + tid < len)
       partialSum[tid] = input[start + tid];
    else
       partialSum[tid] = 0;

    if (start + SUM_BLOCK_SIZE + tid < len)
       partialSum[SUM_BLOCK_SIZE + tid] = input[start + SUM_BLOCK_SIZE + tid];
    else
       partialSum[SUM_BLOCK_SIZE + tid] = 0;

    //Traverse the reduction tree
    for (int stride = SUM_BLOCK_SIZE; stride >= 1; stride >>= 1) {
       __syncthreads();
       if (tid < stride)
          partialSum[tid] += partialSum[tid+stride];
    }
    //Write the computed sum of the block to the output vector at the correct index
    if (tid == 0)
       output[blockIdx.x] = partialSum[0];
}

length_t cuStinger::sumDeviceArray(length_t* arr, length_t len){
	length_t numOutputElements = len / (SUM_BLOCK_SIZE<<1);
    if (len % (SUM_BLOCK_SIZE<<1)) {
        numOutputElements++;
    }

	length_t* d_out = (length_t*)allocDeviceArray(len, sizeof(length_t*));

	devSumArray<<<numOutputElements,SUM_BLOCK_SIZE>>>(arr,d_out,len);

	length_t* h_out = (length_t*)allocHostArray(len, sizeof(length_t*));
	
	length_t sum=0;
	copyArrayDeviceToHost(d_out, h_out, len, sizeof(length_t));
	for(int i=0; i<numOutputElements; i++){
		sum+=h_out[i];
	}
	freeHostArray(h_out);
	freeDeviceArray(d_out);	
	return sum;
}

__global__ void deviceCopyMultipleAdjacencies(cuStinger* custing, cuStinger::cusVertexData* olddVD, 
	vertexId_t* requireUpdates, length_t requireCount ,length_t verticesPerThreadBlock)
{
	// int32_t** d_cuadj = custing->d_adj;
	// length_t* d_utilized = custing->getDeviceUsed();

	length_t v_init=blockIdx.x*verticesPerThreadBlock;
	for (int v_hat=0; v_hat<verticesPerThreadBlock; v_hat++){
		if((v_init+v_hat)>=requireCount)
			break;
		vertexId_t v=requireUpdates[v_init+v_hat];

		cuStinger::cusEdgeData *dED = custing->dVD->adj[v];
		cuStinger::cusEdgeData *olddED = olddVD->adj[v];

		//epv = edge per vertex
		length_t epv = olddVD->getMax()[v];
		int32_t pos=0;

		dED->mem = custing->dVD->edMem[v];
		dED->dst = (vertexId_t*)(dED->getMem() + pos); 	pos+=sizeof(vertexId_t)*epv;
		dED->ew  = (eweight_t*)(dED->getMem() + pos); 	pos+=sizeof(eweight_t)*epv;
		dED->et  = (etype_t*)(dED->getMem() + pos); 	pos+=sizeof(etype_t)*epv;
		dED->t1  = (timestamp_t*)(dED->getMem() + pos); pos+=sizeof(timestamp_t)*epv;
		dED->t2  = (timestamp_t*)(dED->getMem() + pos); pos+=sizeof(timestamp_t)*epv;

		__syncthreads();

		for(length_t e=threadIdx.x; e<olddVD->getUsed()[v]; e+=blockDim.x){
			dED->dst[e] = olddED->dst[e];
			if(custing->isSemantic){
				dED->ew[e] = olddED->ew[e];	
			}
			else if(custing->useEWeight){
				dED->ew[e] = olddED->ew[e];	
				dED->et[e] = olddED->et[e];	
				dED->t1[e] = olddED->t1[e];	
				dED->t2[e] = olddED->t1[e];	
			}
		}
	}
}


void cuStinger::copyMultipleAdjacencies(cusVertexData* olddVD, 
	vertexId_t* requireUpdates, length_t requireCount){

	dim3 numBlocks(1, 1);
	int32_t threads=32;
	dim3 threadsPerBlock(threads, 1);

	numBlocks.x = ceil((float)requireCount);
	if (numBlocks.x>16000){
		numBlocks.x=16000;
	}	
	int32_t verticesPerThreadBlock;
	if(numBlocks.x == requireCount)
		verticesPerThreadBlock=1;
	else
		verticesPerThreadBlock = ceil(float(requireCount)/float(numBlocks.x-1));


	deviceCopyMultipleAdjacencies<<<numBlocks,threadsPerBlock>>>(d_cuStinger,
		olddVD, requireUpdates, requireCount, verticesPerThreadBlock);
	checkLastCudaError("Error in the first update sweep");
}



__global__ void deviceCheckForDuplicateEdges(cuStinger* custing, length_t verticesPerThreadBlock)
{
	vertexId_t v_init=blockIdx.x*verticesPerThreadBlock+threadIdx.x;
	length_t nv = custing->getMaxNV();

	__shared__ int dupFound;

	for (vertexId_t v_hat=0; v_hat<verticesPerThreadBlock; v_hat++){
		vertexId_t v=v_init+v_hat;
		if(v>=nv)
			break;
		length_t edges = custing->dVD->getUsed()[v];
		cuStinger::cusEdgeData *dED = custing->dVD->adj[v];

		// if(v ==45788 && threadIdx.x==0){
		// 	for(length_t e=0; e<edges; e++)
		// 		printf("%d ,",dED->dst[e]);
		// 	printf("\n");
		// }

		for (length_t e=0; e<edges; e++){
			vertexId_t currDest=dED->dst[e];
			dupFound=-1;
			__syncthreads();


			for (length_t e2=0; e2<edges; e2+=blockDim.x){
				vertexId_t currDest2 = dED->dst[e2];
				if(currDest==currDest2 && e!=e2){
					dupFound=e2;
				}
			}
			__syncthreads();
			if(dupFound!=-1)
				printf("DUP FOUND IN CUSTIGER\n");
		}

	}


}


void cuStinger::checkDuplicateEdges(){

	dim3 numBlocks(1, 1);
	int32_t threads=32;
	dim3 threadsPerBlock(threads, 1);
	int32_t verticesPerThreadBlock;

	numBlocks.x = ceil((float)nv/(float)threads);
	if (numBlocks.x>16000){
		numBlocks.x=16000;
	}	
	verticesPerThreadBlock = ceil(float(nv)/float(numBlocks.x));

	// cout << "checkDuplicateEdges : " << verticesPerThreadBlock<< endl;
	// cout << "checkDuplicateEdges : " << numBlocks.x << endl;
	// cout << "Deletions : " << threadsPerBlock.x << endl;

	deviceCheckForDuplicateEdges<<<numBlocks,threadsPerBlock>>>(d_cuStinger,
		verticesPerThreadBlock);
	checkLastCudaError("Error in the first update sweep");
}




