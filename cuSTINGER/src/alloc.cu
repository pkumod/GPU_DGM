
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <stdio.h>
#include <string.h>

#include "utils.hpp"

using namespace std;


void* allocHostArray(length_t elements,int32_t eleSize){
	if (elements==0 || eleSize==0)
		return NULL;
	return malloc(eleSize*elements);
}

void* allocDeviceArray(length_t elements,int32_t eleSize){
	int32_t* ptr=NULL;
	if (elements==0 || eleSize==0)
		return NULL;
	checkCudaErrors (hipMalloc((void **)&ptr,eleSize*elements));
	return ptr;
}

void freeHostArray(void* array){
	free(array);
}

void freeDeviceArray(void* array){
	checkCudaErrors(hipFree(array));
}


void copyArrayHostToHost(void* hostSrc,  void* hostDst, length_t elements, int32_t eleSize){
	memcpy(hostDst,hostSrc,elements*eleSize);
}

void copyArrayHostToDevice(void* hostSrc, void* devDst, length_t elements, int32_t eleSize){
	checkCudaErrors(hipMemcpy(devDst,hostSrc,elements*eleSize,hipMemcpyHostToDevice));
}

void copyArrayDeviceToHost(void* devSrc, void* hostDst, length_t elements, int32_t eleSize){
	checkCudaErrors(hipMemcpy(hostDst,devSrc,elements*eleSize,hipMemcpyDeviceToHost));
}

void copyArrayDeviceToDevice(void* devSrc, void* devDst, length_t elements, int32_t eleSize){
	checkCudaErrors(hipMemcpy(devDst,devSrc,elements*eleSize,hipMemcpyDeviceToDevice));
}


//Note: Times are returned in seconds
void start_clock(hipEvent_t &start, hipEvent_t &end)
{
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&end));
    checkCudaErrors(hipEventRecord(start,0));
}

float end_clock(hipEvent_t &start, hipEvent_t &end)
{
    float time;
    checkCudaErrors(hipEventRecord(end,0));
    checkCudaErrors(hipEventSynchronize(end));
    checkCudaErrors(hipEventElapsedTime(&time,start,end));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(end));

    return time/(float)1000;
}

