#include "rpma.cuh"
#include "util.cuh"
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <utility>
#include <cstring>

using namespace std;

vector<string> datasets;

bool check_csr_result(vector<vector<pair<SIZE_TYPE, SIZE_TYPE> > > &G, DEV_VEC_SIZE &d_row_offset, DEV_VEC_SIZE &d_col_index) {

	thrust::host_vector<SIZE_TYPE> row_offset(d_row_offset.size()), col_index(d_col_index.size());
	cErr(hipMemcpy(RAW_PTR(row_offset), RAW_PTR(d_row_offset), d_row_offset.size() * sizeof(SIZE_TYPE), hipMemcpyDeviceToHost));
	cErr(hipMemcpy(RAW_PTR(col_index), RAW_PTR(d_col_index), d_col_index.size() * sizeof(SIZE_TYPE), hipMemcpyDeviceToHost));

	assert(G.size() + 1 == row_offset.size());

	for (int i = 0; i < G.size(); ++i) {
		SIZE_TYPE nbrlist_len = row_offset[i + 1] - row_offset[i] - 1;
		if (G[i].size() != nbrlist_len) {
			cout << " src " << i << ": size doesnt match" << endl;
			cout << "got " << nbrlist_len << ", but expect " << G[i].size() << endl;
			return false;
		}
		for (int j = 0; j < nbrlist_len; ++j) {
			if (col_index[row_offset[i] + j] != G[i][j].first) {
				cout << " src " << i << ": nbr doesnt match:" << j << endl;
				cout << "got " << col_index[row_offset[i] + j] << ", but expect " << G[i][j].first << endl;
				return false;
			}
		}
		assert(col_index[row_offset[i] + G[i].size()] == i);
	}
	return true;
}


bool
test_csr_rpma(thrust::host_vector<KEY_TYPE> &h_keys, \
	vector<vector<pair<SIZE_TYPE, SIZE_TYPE> > > &G, SIZE_TYPE node_num, SIZE_TYPE edge_num) {

	RPMA rpma;
	init_csr_rpma(rpma, node_num);

	DEV_VEC_KEY keys(edge_num);
	DEV_VEC_VALUE values(edge_num);
	cErr(hipMemcpy(RAW_PTR(keys), RAW_PTR(h_keys), sizeof(KEY_TYPE) * edge_num, hipMemcpyHostToDevice));
	cErr(hipMemset(RAW_PTR(values), 0, sizeof(VALUE_TYPE) * edge_num));
	cErr(hipDeviceSynchronize());
		
	update_rpma(rpma, keys, values, 0);
	cErr(hipDeviceSynchronize());

TimeKeeper t2;
cErr(hipDeviceSynchronize());
	DEV_VEC_SIZE row_offset, col_index;
	to_csr_rpma(rpma, row_offset, col_index);
cErr(hipDeviceSynchronize());
cout << t2.checkTime("") << endl;

	return check_csr_result(G, row_offset, col_index);
}

void
read_edgelist_and_timestamps(string &filename, thrust::host_vector<KEY_TYPE> &h_keys, \
	vector<vector<pair<SIZE_TYPE, SIZE_TYPE> > > &G, SIZE_TYPE &node_num, SIZE_TYPE &edge_num, SIZE_TYPE edge_n) {
	ifstream e_file, t_file;
	e_file.open((filename + ".edgelist").c_str());
	t_file.open((filename + ".timestamps").c_str());

	e_file >> node_num;
	e_file >> edge_num;
	if (edge_n)
		edge_num = edge_n;

	G.resize(node_num);
	for (int i = 0; i < edge_num; ++i) {
		KEY_TYPE src, dst, time;
		e_file >> src;
		e_file >> dst;
		t_file >> time;

		assert(src < node_num && dst < node_num);
		G[src].push_back(make_pair(dst, time));
		h_keys.push_back((src << SRC_SHIFT)| (dst << DST_SHIFT) | time);
	}
	

	for (int i = 0; i < node_num; ++i) {
		sort(G[i].begin(), G[i].end());
	}

	e_file.close();
	t_file.close();
}

void
test_all(string dataset, SIZE_TYPE edge_n) {
	cout << "reading edgelist and timestamps" << endl;
	SIZE_TYPE node_num, edge_num;
	thrust::host_vector<KEY_TYPE> h_keys;
	vector<vector<pair<SIZE_TYPE, SIZE_TYPE> > > G;
	read_edgelist_and_timestamps(dataset, h_keys, G, node_num, edge_num, edge_n);
	cout << "node num is:" << node_num << ", edge num is:" << edge_num << endl;

	if (test_csr_rpma(h_keys, G, node_num, edge_num))
		cout << "csr test success" << endl;
	else {
		cout << "csr test fail" << endl;
		return;
	}
}


int
main(int args, char *argv[]) {

	SIZE_TYPE edge_num = atoi(argv[1]);

	cErr(hipDeviceSynchronize());
	cErr(hipDeviceSetLimit(hipLimitMallocHeapSize, 1048*1024*1024));
	cErr(hipDeviceSynchronize());

	datasets.push_back(argv[2]);
	//datasets.push_back("livejournal");
	//datasets.push_back("graph500");
	for (auto dataset = datasets.begin(); dataset != datasets.end(); ++dataset) {
		cout << "testing " << *dataset << endl;
		test_all(string("../edgelists/") + *dataset, edge_num);
	}
	return 0;
}
