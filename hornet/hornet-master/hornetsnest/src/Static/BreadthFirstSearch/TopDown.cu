#include "hip/hip_runtime.h"
/**
 * @author Federico Busato                                                  <br>
 *         Univerity of Verona, Dept. of Computer Science                   <br>
 *         federico.busato@univr.it
 * @date September, 2017
 * @version v2
 *
 * @copyright Copyright © 2017 Hornet. All rights reserved.
 *
 * @license{<blockquote>
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * * Neither the name of the copyright holder nor the names of its
 *   contributors may be used to endorse or promote products derived from
 *   this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * </blockquote>}
 */
#include "Static/BreadthFirstSearch/TopDown.cuh"
#include <Graph/BFS.hpp>
#include <Graph/GraphStd.hpp>

namespace hornets_nest {

const dist_t INF = std::numeric_limits<dist_t>::max();

//------------------------------------------------------------------------------
///////////////
// OPERATORS //
///////////////

struct BFSOperator {
    dist_t* __restrict__ d_distances;
    dist_t               current_level;
    TwoLevelQueue<vert_t> queue;

    OPERATOR(Vertex& vertex, Edge& edge) {
        auto dst = edge.dst_id();
        if (d_distances[dst] == INF) {
            d_distances[dst] = current_level;
            queue.insert(dst);
        }
    }
};

//------------------------------------------------------------------------------
/////////////////
// BfsTopDown //
/////////////////

BfsTopDown::BfsTopDown(HornetGraph& hornet) :
                                 StaticAlgorithm(hornet),
                                 queue(hornet),
                                 load_balancing(hornet) {
    gpu::allocate(d_distances, hornet.nV());
    reset();
}

BfsTopDown::~BfsTopDown() {
    gpu::free(d_distances);
}

void BfsTopDown::reset() {
    current_level = 1;
    queue.clear();

    auto distances = d_distances;
    forAllnumV(hornet, [=] __device__ (int i){ distances[i] = INF; } );
}

void BfsTopDown::set_parameters(vert_t source) {
    bfs_source = source;
    queue.insert(bfs_source);                   // insert source in the frontier
    gpu::memsetZero(d_distances + bfs_source);  // reset source distance
}

void BfsTopDown::run() {
    while (queue.size() > 0) {
        //std::cout << queue.size() << std::endl;
        //for all edges in "queue" applies the operator "BFSOperator" by using
        //the load balancing algorithm instantiated in "load_balancing"
        forAllEdges(hornet, queue,
                    BFSOperator { d_distances, current_level, queue },
                    load_balancing);
        //todo: gpu::forAllEdges
        current_level++;
        queue.swap();
    }
}

void BfsTopDown::release() {
    gpu::free(d_distances);
    d_distances = nullptr;
}

bool BfsTopDown::validate() {
    //std::cout << "\nTotal enqueue vertices: "
    //          << xlib::format(queue.enqueue_items())
    //          << std::endl;

    //using namespace graph;
    //GraphStd<vert_t, eoff_t> graph(hornet.csr_offsets(), hornet.nV(),
    //                              hornet.csr_edges(), hornet.nE());
    //BFS<vert_t, eoff_t> bfs(graph);
    //bfs.run(bfs_source);

    //auto h_distances = bfs.result();
    //return gpu::equal(h_distances, h_distances + graph.nV(), d_distances);
    //TODO : Create GraphStd from hornet class
    return true;
}

} // namespace hornets_nest
