
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int num_gpus;
    if (hipGetDeviceCount(&num_gpus) != hipSuccess)
        return EXIT_FAILURE;

    std::cout << num_gpus << ";";
    for (int i = 0; i < num_gpus; i++) {
        hipDeviceProp_t devive_prop;
        if (hipGetDeviceProperties(&devive_prop, i) != hipSuccess)
            return EXIT_FAILURE;
        std::cout << devive_prop.major * 10 + devive_prop.minor << ";"
                  << devive_prop.multiProcessorCount            << ";"
                  << devive_prop.name;
        if (i < num_gpus - 1)
            std::cout << ";";
    }
}
